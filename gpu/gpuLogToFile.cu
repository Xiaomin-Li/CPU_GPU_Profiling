/*
  Copyright (c) 2013, Texas State University-San Marcos. All rights reserved.

  Redistribution and use in source and binary forms, with or without modification,
  are permitted for academic, research, experimental, or personal use provided
  that the following conditions are met:

  * Redistributions of source code must retain the above copyright notice,
  this list of conditions and the following disclaimer.
  * Redistributions in binary form must reproduce the above copyright notice,
  this list of conditions and the following disclaimer in the documentation
  and/or other materials provided with the distribution.
  * Neither the name of Texas State University-San Marcos nor the names of its
  contributors may be used to endorse or promote products derived from this
  software without specific prior written permission.

  For all other uses, please contact the Office for Commercialization and Industry
  Relations at Texas State University-San Marcos <http://www.txstate.edu/ocir/>.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
  DISCLAIMED IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
  ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
  ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

  Author: Martin Burtscher (in collaboration with Ivan Zecena and Ziliang Zong)
*/

//compile 
//nvcc -I../include -O3 -w gpuLogToFile.cu -o gpuToFile -L/usr/lib64/nvidia -lnvidia-ml
//only used for rainbow-panda server which has four devices 


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/types.h>
#include <signal.h>
#include "nvml.h"

#define DEVICE0 0
#define DEVICE1 1
#define DEVICE2 2
#define DEVICE3 3

double secondsSince(struct timeval *startTime) {
    struct timeval currentTime;
    gettimeofday(&currentTime, NULL);
    return ((currentTime.tv_sec*1e6 + currentTime.tv_usec) - (startTime->tv_sec*1e6 + startTime->tv_usec)) / 1e6;
}


static inline double getTime()
{
    struct timeval time;
    gettimeofday(&time, NULL);
    return time.tv_sec + time.tv_usec * 0.000001;
}

static void initAndTest(nvmlDevice_t *device0, nvmlDevice_t *device1, nvmlDevice_t *device2, nvmlDevice_t *device3)
{
    nvmlReturn_t result;
    nvmlMemory_t mem;
    unsigned int power;

    result = nvmlInit();
    if (NVML_SUCCESS != result) {
        printf("failed to initialize NVML: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetHandleByIndex(DEVICE0, device0);
    if (NVML_SUCCESS != result) {
        printf("failed to get handle for device: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetHandleByIndex(DEVICE1, device1);
    if (NVML_SUCCESS != result) {
        printf("failed to get handle for device: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetHandleByIndex(DEVICE2, device2);
    if (NVML_SUCCESS != result) {
        printf("failed to get handle for device: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetHandleByIndex(DEVICE3, device3);
    if (NVML_SUCCESS != result) {
        printf("failed to get handle for device: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetPowerUsage(*device0, &power);
    if (NVML_SUCCESS != result) {
        printf("failed to read power: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetPowerUsage(*device1, &power);
    if (NVML_SUCCESS != result) {
        printf("failed to read power: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetPowerUsage(*device2, &power);
    if (NVML_SUCCESS != result) {
        printf("failed to read power: %s\n", nvmlErrorString(result));
        exit(1);
    }

    result = nvmlDeviceGetPowerUsage(*device3, &power);
    if (NVML_SUCCESS != result) {
        printf("failed to read power: %s\n", nvmlErrorString(result));
        exit(1);
    }
}

static inline void getInfo
    (
        nvmlDevice_t device0, nvmlDevice_t device1, nvmlDevice_t device2, nvmlDevice_t device3, 
        unsigned int *power0, unsigned int *power1, unsigned int *power2, unsigned int *power3, 
        unsigned int *temp0, unsigned int *temp1, unsigned int *temp2, unsigned int *temp3,
        nvmlUtilization_t *u0, nvmlUtilization_t *u1, nvmlUtilization_t *u2, nvmlUtilization_t *u3,
        FILE* outputFile, struct timeval *startTime
    )
{

    nvmlDeviceGetPowerUsage(device0, power0);
    *power0 *= .001;

    nvmlDeviceGetPowerUsage(device1, power1);
    *power1 *= .001;

    nvmlDeviceGetPowerUsage(device2, power2);
    *power2 *= .001;

    nvmlDeviceGetPowerUsage(device3, power3);
    *power3 *= .001;


    nvmlDeviceGetTemperature(device0, NVML_TEMPERATURE_GPU, temp0);
    nvmlDeviceGetTemperature(device1, NVML_TEMPERATURE_GPU, temp1);
    nvmlDeviceGetTemperature(device2, NVML_TEMPERATURE_GPU, temp2);
    nvmlDeviceGetTemperature(device3, NVML_TEMPERATURE_GPU, temp3);

    nvmlDeviceGetUtilizationRates(device0, u0);
    nvmlDeviceGetUtilizationRates(device1, u1);
    nvmlDeviceGetUtilizationRates(device2, u2);
    nvmlDeviceGetUtilizationRates(device3, u3);

    unsigned int total_power;
    total_power = *power0 + *power1 + *power2 + *power3;

    struct timeval currentTime;
    double time_interval;
    gettimeofday(&currentTime, NULL);
    //get device utilization api only support fermi and quadro architrcture cards.
    //for panda server, it contains RTX 2080ti cards which don't have supportion.
    //util->gpu return gpu utilization, util->memory return gpu memory utilization  
    time_interval = ((currentTime.tv_sec*1e6 + currentTime.tv_usec) - (startTime->tv_sec*1e6 + startTime->tv_usec)) / 1e6;
    fprintf(outputFile, "%f, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u, %u\n", 
            secondsSince(startTime), total_power, 
            *power0, *temp0, u0->gpu, *power1, *temp1, u1->gpu, *power2, *temp2, u2->gpu, *power3, *temp3, u3->gpu 
            );
}

static void sigterm_hdl(int sig) {
    nvmlShutdown();
    exit(1);
}

int main(int argc, char *argv[])
{
    nvmlDevice_t device0, device1, device2, device3;
    unsigned int power0, power1, power2, power3, delay_us;
    unsigned int temp0, temp1, temp2, temp3;
    nvmlUtilization_t u0, u1, u2, u3;


    if (argc != 3 || atoi(argv[1]) <= 0) {
        fprintf(stderr, "Usage: %s [sampling rate (Hz)] [output filename]\n", argv[0]);
        return 1;
    }
	delay_us = 1e6 / atoi(argv[1]);
	char filename[512];
	char hostname[9];
	hostname[8] = NULL;
	gethostname(hostname, 8);
	snprintf(filename, 512, "%s_GPU-%s.csv", hostname, argv[2]);
	FILE *outputFile = fopen(filename, "w");
	if (outputFile == NULL) {
	   fprintf(stderr, "Unable to open output file.\n");
	   return 1;
	}
	setbuf(outputFile, NULL);
    if (delay_us <= 0) {
        fprintf(stderr, "[GPU meter]: Sampling delay must be a nonnegative integer.");
        return 1;
    }

    // SIGTERM handler
    struct sigaction sa;
    memset(&sa, 0, sizeof(sa));
    sa.sa_handler = sigterm_hdl;
    if (sigaction(SIGTERM, &sa, 0)) {
        fprintf(stderr,"[GPU meter]: Sigaction failed.\n");
        exit(1);
    }

    initAndTest(&device0, &device1, &device2, &device3);

    // We write this 'Y' to STDOUT so master_meter will know that we're ready to start logging.
    // The master meter will block until this has been read. 
    char c = 'Y';
    write(STDOUT_FILENO, &c, 1);

	fprintf(outputFile, "Time(S), Total(w), power0(W), temp0(C), util0, power1(W), temp1(C), util1, power2(W), temp2(C), util1, power3(W), temp3(C), util3\n");
    // Begin power measurement.
	struct timeval start;
	gettimeofday(&start, NULL);
    do {
        usleep(delay_us);
        getInfo
            (
                device0, device1, device2, device3, 
                &power0, &power1, &power2, &power3, 
                &temp0, &temp1, &temp2, &temp3,
                &u0, &u1, &u2, &u3,
                outputFile, &start
            );
    } while(1);
}